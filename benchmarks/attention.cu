#include "hip/hip_runtime.h"
// nvcc -std=c++17 -Xptxas=-v -lineinfo -O3 --use_fast_math -gencode=arch=compute_80,code=sm_80 -I ../csrc/ -I ../third_party/cutlass/include --disable-warnings -o attention attention.cu
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#include<thrust/device_vector.h>
#include<thrust/random.h>
#include<thrust/transform.h>
#include<thrust/iterator/counting_iterator.h>

#include <iostream>
#include <vector>

#include "kernels/attention/smemfuse/nnfusion_rt.h"
#include "kernels/attention/regfuse/nnfusion_rt.h"

/*
constexpr int Br = {Br};
constexpr int Bc = {Bc};
constexpr int Kd = {Kd};
constexpr int D = {D};

constexpr bool unrollLastIter = {unrollLastIter};
// for q&k splitk
__device__ constexpr int BlockKSmem = {BlockKSmem};
constexpr int num_stages_qk = {num_stages_qk};
constexpr bool load_q_once = (BlockKSmem == Kd);
// for V splitk
constexpr int BlockKSmem2 = {BlockKSmem2};
constexpr int num_stages_v = {num_stages_v};
// for sQ,sK,sV,sO swizzle
constexpr int SmemKAtom = BlockKSmem % 64 == 0 ? 64 : 32;
constexpr int kSwizzle = SmemKAtom == 32 ? 2 : 3;

constexpr int shared_matmulqkv = num_stages_qk*(Br)*BlockKSmem*sizeof(half)+num_stages_qk*Bc*BlockKSmem*sizeof(half)+num_stages_v*BlockKSmem2*D*sizeof(half);
constexpr int shared_out = Br * D * sizeof(half);
constexpr int shared_mem = (shared_matmulqkv) > shared_out ? (shared_matmulqkv):shared_out;//(acc_o(p(q,k),v))

constexpr int Nthreads = {Nthreads};*/


class ProblemShape{
public:
    ProblemShape(int batch,int head,int seqlen_q,int seqlen_kv):B(batch),H(head),Seq_q(seqlen_q),Seq_k(seqlen_kv){};

    int B,H,Seq_q,Seq_k;
};

template<int Br_, int Bc_,int Kd_, int D_, int Nthreads_, 
/*smem_fuse only*/ int warps_mma1_N_ = 1, int warps_mma_N_ = 1, 
int BlockKSmem_=Kd_, int num_stages_qk_=1, int BlockKSmem2_=Bc_, int num_stages_v_=1, int SmemKAtom_=64, bool unrollLastIter_=true, 
/*smem_fuse only*/int SmemKAtomV_ = 64>
class ImplementShape{
public:
    constexpr static int Br = Br_;
    constexpr static int Bc = Bc_;
    constexpr static int Kd = Kd_;
    constexpr static int D = D_;
    constexpr static int Nthreads = Nthreads_;
    constexpr static int BlockKSmem = BlockKSmem_;
    constexpr static int num_stages_qk = num_stages_qk_;
    constexpr static bool load_q_once = (BlockKSmem == Kd);
    constexpr static int BlockKSmem2 = BlockKSmem2_;
    constexpr static int num_stages_v = num_stages_v_;
    constexpr static int SmemKAtom = SmemKAtom_;
    constexpr static int kSwizzle = SmemKAtom == 32 ? 2 : 3;
    constexpr static bool unrollLastIter = unrollLastIter_;

    constexpr static int SmemKAtomV = SmemKAtomV_;
    constexpr static int kSwizzleV = SmemKAtomV == 32 ? 2 : 3;
    constexpr static int SmemKAtomP = Bc % 64 == 0 ? 64 : 32;
    constexpr static int kSwizzleP = SmemKAtomP == 32 ? 2 : 3;
    constexpr static int SmemKAtomPf16 = 64;
    constexpr static int kSwizzlePf16 = SmemKAtomPf16 == 32 ? 2 : 3;
    constexpr static int warps_mma1_N = warps_mma1_N_;
    constexpr static int warps_mma_N = warps_mma_N_;
};

struct prg
{
    float a,b;
    __host__ __device__ 
    prg(float _a=0.f, float _b=1.f):a(_a),b(_b){};
    __host__ __device__ half operator()(const unsigned int n){
        thrust::default_random_engine rng;
        thrust::normal_distribution<float> dist(a,b);
        rng.discard(n);

        return half(dist(rng));
    }
};

template <typename InplementConfig>
float test_regfuse_attention(ProblemShape shape){
    constexpr int br = InplementConfig::Br;
    constexpr int bc = InplementConfig::Bc;
    constexpr int kd = InplementConfig::Kd;
    constexpr int d = InplementConfig::D;
    constexpr int Nthreads = InplementConfig::Nthreads;
    constexpr int BlockKSmem = InplementConfig::BlockKSmem;
    constexpr int num_stages_qk = InplementConfig::num_stages_qk;
    constexpr bool load_q_once = InplementConfig::load_q_once;
    constexpr int BlockKSmem2 = InplementConfig::BlockKSmem2;
    constexpr int num_stages_v = InplementConfig::num_stages_v;
    constexpr int SmemKAtom = InplementConfig::SmemKAtom;
    constexpr int kSwizzle = InplementConfig::kSwizzle;
    constexpr bool unrollLastIter = InplementConfig::unrollLastIter;

    int B = shape.B;
    int H = shape.H;
    int Seq_q = shape.Seq_q;
    int Seq_k = shape.Seq_k;

    int shared_matmulqkv = num_stages_qk*(br)*BlockKSmem*sizeof(half)+num_stages_qk*bc*BlockKSmem*sizeof(half)+num_stages_v*BlockKSmem2* d* sizeof(half);
    int shared_out = br * d * sizeof(half);
    int shared_mem = (shared_matmulqkv) > shared_out ? (shared_matmulqkv):shared_out;//(acc_o(p(q,k),v))

    auto kernel = &flashattn_fwd_regfuse<kd,d,br,bc,Nthreads,BlockKSmem,num_stages_qk,load_q_once,BlockKSmem2,num_stages_v,SmemKAtom,kSwizzle,unrollLastIter>;
    if(shared_mem > 48*1024){
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem);
    }

    thrust::device_vector<int> cache(int(256e6/4));
    //input argument
    thrust::device_vector<half> Parameter_0_0_0(B*H*Seq_q*kd);
    thrust::device_vector<half> Parameter_1_0_0(B*H*Seq_k*kd);
    thrust::device_vector<half> Parameter_2_0_0(B*H*Seq_k*d);
    //output argument
    thrust::device_vector<half> Result_7_0_0(B*H*Seq_q*d);
    thrust::counting_iterator<unsigned int>  index_begin(0);
    thrust::transform(index_begin, index_begin + B*H*Seq_q*kd, Parameter_0_0_0.begin(), prg());
    thrust::transform(index_begin, index_begin + B*H*Seq_k*kd, Parameter_1_0_0.begin(), prg());
    thrust::transform(index_begin, index_begin + B*H*Seq_k*d, Parameter_2_0_0.begin(), prg());
    
    auto q_ptr = thrust::raw_pointer_cast(Parameter_0_0_0.data());
    auto k_ptr = thrust::raw_pointer_cast(Parameter_1_0_0.data());
    auto v_ptr = thrust::raw_pointer_cast(Parameter_2_0_0.data());
    auto o_ptr = thrust::raw_pointer_cast(Result_7_0_0.data());

    float ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
    hipEventRecord(start, 0);
    for(int _ = 0; _ < 5; _++)
        kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
    if(hipEventRecord(stop, 0) != hipSuccess) return -1;
    if(hipEventSynchronize(stop) != hipSuccess) return -1;
    if(hipGetLastError() != hipSuccess) {
        printf("CUDA error: %s\\n", hipGetErrorString(hipGetLastError()));
        return -1;
    }
    hipEventElapsedTime(&ms, start, stop);
    int warm_up = int(ceil(50.0 / (ms/5)));
    int repeats = int(ceil(100.0 / (ms/5)));
    for(int _ = 0; _ < warm_up; _++){
        kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
    }

    std::vector<hipEvent_t> start_(repeats);
    std::vector<hipEvent_t> stop_(repeats);
    for(int ii = 0; ii < repeats; ii++){
        hipEventCreate(&start_[ii]);
        hipEventCreate(&stop_[ii]);
    }
    for(int ii = 0; ii < repeats; ii++){
        thrust::fill(cache.begin(), cache.end(), ii);
        hipEventRecord(start_[ii], 0);
        kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
        hipEventRecord(stop_[ii], 0);
    }
    if(hipEventSynchronize(stop_[repeats-1]) != hipSuccess) return -1;
    if(hipGetLastError() != hipSuccess) {
        printf("CUDA error: %s\\n", hipGetErrorString(hipGetLastError()));
        return -1;
    }
    ms = 0;
    for(int ii = 0; ii < repeats; ii++){
        float tmp;
        hipEventElapsedTime(&tmp, start_[ii], stop_[ii]);
        ms += tmp;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop); 
    for(int ii = 0; ii < repeats; ii++){
        hipEventDestroy(start_[ii]);
        hipEventDestroy(stop_[ii]);
    }
    return ms / repeats;

}

template <typename InplementConfig>
float test_smemfuse_attention(ProblemShape shape){
    constexpr int br = InplementConfig::Br;
    constexpr int bc = InplementConfig::Bc;
    constexpr int kd = InplementConfig::Kd;
    constexpr int d = InplementConfig::D;
    constexpr int Nthreads = InplementConfig::Nthreads;
    constexpr int BlockKSmem = InplementConfig::BlockKSmem;
    constexpr int num_stages_qk = InplementConfig::num_stages_qk;
    constexpr bool load_q_once = InplementConfig::load_q_once;
    constexpr int BlockKSmem2 = InplementConfig::BlockKSmem2;
    constexpr int num_stages_v = InplementConfig::num_stages_v;
    constexpr int SmemKAtom = InplementConfig::SmemKAtom;
    constexpr int kSwizzle = InplementConfig::kSwizzle;
    constexpr bool unrollLastIter = InplementConfig::unrollLastIter;

    constexpr int SmemKAtomV = InplementConfig::SmemKAtomV;
    constexpr int kSwizzleV = InplementConfig::kSwizzleV;
    constexpr int SmemKAtomP = InplementConfig::SmemKAtomP;
    constexpr int kSwizzleP = InplementConfig::kSwizzleP;
    constexpr int SmemKAtomPf16 = InplementConfig::SmemKAtomPf16;
    constexpr int kSwizzlePf16 = InplementConfig::kSwizzlePf16;
    constexpr int warps_mma1_N = InplementConfig::warps_mma1_N;
    constexpr int warps_mma_N = InplementConfig::warps_mma_N;


    int B = shape.B;
    int H = shape.H;
    int Seq_q = shape.Seq_q;
    int Seq_k = shape.Seq_k;

    int shared_matmulqkv = num_stages_qk*(br)*BlockKSmem*sizeof(half)+num_stages_qk*bc*BlockKSmem*sizeof(half)+num_stages_v*BlockKSmem2* d* sizeof(half);
    int shared_accs = br*bc*sizeof(float)+br*bc*sizeof(half) + 3*sizeof(float)*br;
    int shared_out = br * d * sizeof(half);
    int shared_mem = (shared_matmulqkv+shared_accs) > shared_out ? (shared_matmulqkv+shared_accs):shared_out;//(acc_o(p(q,k),v))

    auto kernel = &flashattn_fwd_smemfuse<kd,d,br,bc,Nthreads,BlockKSmem,num_stages_qk,load_q_once,BlockKSmem2,num_stages_v,SmemKAtom,kSwizzle,SmemKAtomV,kSwizzleV,SmemKAtomP,kSwizzleP,SmemKAtomPf16,kSwizzlePf16,warps_mma1_N,warps_mma_N,unrollLastIter>;
    if(shared_mem > 48*1024){
        hipFuncSetAttribute(reinterpret_cast<const void*>(kernel), hipFuncAttributeMaxDynamicSharedMemorySize, shared_mem);
    }

    thrust::device_vector<int> cache(int(256e6/4));
    //input argument
    thrust::device_vector<half> Parameter_0_0_0(B*H*Seq_q*kd);
    thrust::device_vector<half> Parameter_1_0_0(B*H*Seq_k*kd);
    thrust::device_vector<half> Parameter_2_0_0(B*H*Seq_k*d);
    //output argument
    thrust::device_vector<half> Result_7_0_0(B*H*Seq_q*d);
    thrust::counting_iterator<unsigned int>  index_begin(0);
    thrust::transform(index_begin, index_begin + B*H*Seq_q*kd, Parameter_0_0_0.begin(), prg());
    thrust::transform(index_begin, index_begin + B*H*Seq_k*kd, Parameter_1_0_0.begin(), prg());
    thrust::transform(index_begin, index_begin + B*H*Seq_k*d, Parameter_2_0_0.begin(), prg());
    
    auto q_ptr = thrust::raw_pointer_cast(Parameter_0_0_0.data());
    auto k_ptr = thrust::raw_pointer_cast(Parameter_1_0_0.data());
    auto v_ptr = thrust::raw_pointer_cast(Parameter_2_0_0.data());
    auto o_ptr = thrust::raw_pointer_cast(Result_7_0_0.data());

    float ms;
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
    hipEventRecord(start, 0);
    for(int _ = 0; _ < 5; _++)
        kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
    if(hipEventRecord(stop, 0) != hipSuccess) return -1;
    if(hipEventSynchronize(stop) != hipSuccess) return -1;
    if(hipGetLastError() != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
        return -1;
    }
    hipEventElapsedTime(&ms, start, stop);
    int warm_up = int(ceil(50.0 / (ms/5)));
    int repeats = int(ceil(100.0 / (ms/5)));
    for(int _ = 0; _ < warm_up; _++){
        kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
    }

    std::vector<hipEvent_t> start_(repeats);
    std::vector<hipEvent_t> stop_(repeats);
    for(int ii = 0; ii < repeats; ii++){
        hipEventCreate(&start_[ii]);
        hipEventCreate(&stop_[ii]);
    }
    for(int ii = 0; ii < repeats; ii++){
        thrust::fill(cache.begin(), cache.end(), ii);
        hipEventRecord(start_[ii], 0);
        kernel<<<dim3(B*H*Seq_q/br, 1, 1), dim3(Nthreads, 1, 1),shared_mem,0>>>(q_ptr,k_ptr,v_ptr,o_ptr, H,Seq_k,Seq_q);
        hipEventRecord(stop_[ii], 0);
    }
    if(hipEventSynchronize(stop_[repeats-1]) != hipSuccess) return -1;
    if(hipGetLastError() != hipSuccess) {
        printf("CUDA error: %s\n", hipGetErrorString(hipGetLastError()));
        return -1;
    }
    ms = 0;
    for(int ii = 0; ii < repeats; ii++){
        float tmp;
        hipEventElapsedTime(&tmp, start_[ii], stop_[ii]);
        ms += tmp;
    }
    hipEventDestroy(start);
    hipEventDestroy(stop); 
    for(int ii = 0; ii < repeats; ii++){
        hipEventDestroy(start_[ii]);
        hipEventDestroy(stop_[ii]);
    }
    return ms / repeats;

}

int main(){
    ProblemShape PS(4,8,2048,2048);
    using InpleConfig = ImplementShape<128,64,256,256,256>;
    float ms = test_regfuse_attention<InpleConfig>(PS);
    std::cout << "Time: " << ms << "ms" << std::endl;


    ms = test_smemfuse_attention<ImplementShape<64,64,256,256,256,2,4>>(PS);
    std::cout << "Time: " << ms << "ms" << std::endl;
    return 0;
}
